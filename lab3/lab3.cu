#include "hip/hip_runtime.h"
#include "lab3.h"
#include <cstdio>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone(
        const float *background,
        const float *target,
        const float *mask,
        float *output,
        const int wb, const int hb, const int wt, const int ht,
        const int oy, const int ox
        )
{
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;
    const int curt = wt*yt+xt;
    if (yt < ht and xt < wt and mask[curt] > 127.0f) {
        const int yb = oy+yt, xb = ox+xt;
        const int curb = wb*yb+xb;
        if (0 <= yb and yb < hb and 0 <= xb and xb < wb) {
            output[curb*3+0] = target[curt*3+0];
            output[curb*3+1] = target[curt*3+1];
            output[curb*3+2] = target[curt*3+2];
        }
    }
}

__global__ void PoissonImageCloningIteration(const float* fixed, const float* mask, float* input, float* output, int width, int height, int wb, int hb, int ox, int oy) {
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int oneChaDimIdx = (yt * width + xt);
    const int xb = xt + ox;
    const int yb = yt + oy;
    int count = 4;
    if(xb == wb - 1) count -= 1;
    if(xb == 0) count -= 1;
    if(yb == hb - 1) count -= 1;
    if(yb == 0) count -= 1;
    //if(count != 4) printf("nono!\n");
    if(mask[oneChaDimIdx] < 127.0f) return;
    for(int i=0;i<3;i++) {
        int px = oneChaDimIdx * 3 + i;
        float answer = 0.0f;
        if(xt == 0) {
            //no internal
        }
        else if(mask[oneChaDimIdx - 1] > 127.0f) {
            answer += input[px - 3];
        }
        if(xt == width - 1) {
            //no internal
        }
        else if(mask[oneChaDimIdx + 1] > 127.0f) {
            answer += input[px + 3];
        }
        if(yt == 0) {
            //no internal
        }
        else if(mask[oneChaDimIdx - width] > 127.0f) {
            answer += input[px - width * 3];
        }
        if(yt == height - 1) {
            //no internal
        }
        else if(mask[oneChaDimIdx + width] > 127.0f) {
            answer += input[px + width * 3];
        }
        output[px] = (answer + fixed[px]) / count;
    }

}
/*
   __global void JudgeEdge(const float* mask, float* edge, int wt, int ht) {
   const int xt = blockIdx.x * blockDim.x + threadIdx.x;
   const int yt = blockIdx.y * blockDim.y + threadIdx.y;
   const int oneDimIdx = yt * ht + xt;
   if(mask[oneDimIdx] < 127.0f) {
   if(xt != 0) {
   if(mask[oneDimIdx - 1] > 127.0f) {
   edge[oneDimIdx] = 255.0f;
   return;
   }
   }
   else if(xt != wt - 1) {
   if(mask[oneDimIdx + 1] > 127.0f) {
   edge[oneDimIdx] = 255.0f;
   return;
   }
   }
   else if(yt != 0) {
   if(mask[oneDimIdx - wt] > 127.0f) {
   edge[oneDimIdx] = 255.0f;
   return;
   }
   }
   else if(yt != ht - 1) {
   if(mask[oneDimIdx + wt] > 127.0f) {
   edge[oneDimIdx] = 255.0f;
   return;
   }
   }
   }
   edge[oneDimIdx] = 0.0f;
   return;
   }
 */
__global__ void CalculateFixed(const float* background, const float* target, const float* mask, float* fixed, int wb, int hb, int wt, int ht, int oy, int ox) {
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int oneChaDimIdx = (yt * wt + xt);
    const int xb = xt + ox;
    const int yb = yt + oy;
    if(mask[oneChaDimIdx] < 127.0f) return;
    for(int i=0;i<3;i++) {
        int px = oneChaDimIdx * 3 + i;
        int bpx = (yb * wb + xb) * 3 + i;
        float answer = 0.0f;
        //int count = 0;
        if(xt == 0) {
            if(xb != 0) {
                answer += target[px] - 255.0f;
                answer += background[bpx - 3];
            }
            else {
                //no node
            }
        }
        else {
            if(mask[oneChaDimIdx - 1] < 127.0f) {
                if(xb != 0) {
                    answer += target[px] - target[px - 3];
                    answer += background[bpx - 3];
                }
                else{
                    //no node
                }
            }
            else {
                answer += target[px] - target[px - 3];
            }
        }
        if(xt == wt - 1) {
            if(xb != wb - 1) {
                answer += target[px] - 255.0f;
                answer += background[bpx + 3];
            }
            else {
                //no node
            }
        }
        else {
            if(mask[oneChaDimIdx + 1] < 127.0f) {
                if(xb != wb - 1) {
                    answer += target[px] - target[px + 3];
                    answer += background[bpx + 3];
                }
                else {
                    //no node
                }
            }
            else {
                answer += target[px] - target[px + 3];
            }
        }
        if(yt == 0) {
            if(yb != 0) {
                answer += target[px] - 255.0f;
                answer += background[bpx - wb * 3];
            }
            else {
                //no node
            }
        }
        else {
            if(mask[oneChaDimIdx - wt] < 127.0f) {
                if(yb != 0) {
                    answer += target[px] - target[px - wt * 3];
                    answer += background[bpx - wb * 3];
                }
                else {
                    //no node
                }
            }
            else {
                answer += target[px] - target[px - wt * 3];
            }
        }
        if(yt == ht - 1) {
            if(yb != hb - 1) {
                answer += target[px] - 255.0f;
                answer += background[bpx + wb * 3];
            }
            else {
                //no node
            }
        }
        else {
            if(mask[oneChaDimIdx + wt] < 127.0f) {
                if(yb != hb - 1) {
                    answer += target[px] - target[px + wt * 3];
                    answer += background[bpx + wb * 3];
                }
                else {
                    //no node
                }
            }
            else {
                answer += target[px] - target[px + wt * 3];
            }
        }
        fixed[px] = answer;
    }
}

__global__ ImageDownScaleSampling(float* input, float* output, int scale, int wt, int ht) {
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int afterW = wt / scale;
    const int afterH = ht / scale;
    const int px = yt * afterW + xt;
    const int samplePx = (yt * wt + xt) * scale
    output[px] = input[samplePx];
}

__global__ ImageUpScaleInterpolating(float* input, float* output, int nwt, int nht) {
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int px = yt * nwt + xt;
    const int aPx1 = yt * nwt * 4 * xt * 2;
    const int aPx2 = aPx1 + nwt * 2;
    output[aPx1] = input[px];
    output[aPx1 + 1] = input[px];
    output[aPx2] = input[px];
    output[aPx2 + 1] = input[px];
}

void PoissonImageCloning(
        const float *background,
        const float *target,
        const float *mask,
        float *output,
        const int wb, const int hb, const int wt, const int ht,
        const int oy, const int ox
        )
{
    float *fixed, *buf1, *buf2, *tempMask;
    hipMalloc(&fixed, 3*wt*ht*sizeof(float));
    hipMalloc(&buf1, 3*wt*ht*sizeof(float));
    hipMalloc(&buf2, 3*wt*ht*sizeof(float));
    hipMalloc(&tempMask, wt*ht*sizeof(float));

    dim3 gdim8(CeilDiv(wt/8, 32), CeilDiv(ht/8, 16));
    dim3 gdim4(CeilDiv(wt/4, 32), CeilDiv(ht/4, 16));
    dim3 gdim2(CeilDiv(wt/2, 32), CeilDiv(ht/2, 16));
    dim3 gdim(CeilDiv(wt, 32), CeilDiv(ht, 16)), bdim(32, 16);

    ImageDownScaleSampling(mask, tempMask, 8, wt, ht);
    ImageDownScaleSampling(target, buf1, 8, wt, ht);

    CalculateFixed<<<gdim, bdim>>>(
            background, target, mask, fixed,
            wb, hb, wt, ht, oy, ox
            );

    hipMemcpy(buf1, target, sizeof(float)*3*wt*ht, hipMemcpyDeviceToDevice);

    for(int i=0; i<10000; i++) {
        PoissonImageCloningIteration<<<gdim, bdim>>>(
                fixed, mask, buf1, buf2, wt, ht, wb, hb, ox, oy
                );
        PoissonImageCloningIteration<<<gdim, bdim>>>(
                fixed, mask, buf2, buf1, wt, ht, wb, hb, ox, oy
                );
    }



    hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);
    SimpleClone<<<gdim, bdim>>>(
            background, buf1, mask, output,
            wb, hb, wt, ht, oy, ox
            );

    hipFree(fixed);
    hipFree(buf1);
    hipFree(buf2);
}
