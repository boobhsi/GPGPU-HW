#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "../utils/SyncedMemory.h"

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

const int W = 40;
const int H = 12;

__global__ void Draw(char *frame) {
	// TODO: draw more complex things here
	// Do not just submit the original file provided by the TA!
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (y < H and x < W) {
		char c;
		if (x == W-1) {
			c = y == H-1 ? '\0' : '\n';
		} else if (y == 0 or y == H-1 or x == 0 or x == W-2) {
			c = ':';
		} else {
            bool element_drawed = false;
            if (y <= 10 && y >= 5) {
                if (x <= 21 && x >= 8+(10-y)*2) {
                    c = '#';
                    element_drawed = true;
                } else if (x == 33) {
                    if (y == 10) {
                        c = '#';
                        element_drawed = true;
                    } else {
                        c = '|';
                        element_drawed = true;
                    }
                } else if (x == 32 && y == 5) {
                    c = '<';
                    element_drawed = true;
                }
            }
			if (!element_drawed) c = ' ';
		}
		frame[y*W+x] = c;
	}
}

int main(int argc, char **argv)
{
	MemoryBuffer<char> frame(W*H);
	auto frame_smem = frame.CreateSync(W*H);
	CHECK;

	Draw<<<dim3((W-1)/16+1,(H-1)/12+1), dim3(16,12)>>>(frame_smem.get_gpu_wo());
	CHECK;

	puts(frame_smem.get_cpu_ro());
	CHECK;
	return 0;
}
